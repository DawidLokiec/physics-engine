#include "hip/hip_runtime.h"
#include "cudamodule/cuda_acceleration_calculation.h"
#include "physics/astronomical_algorithms.h"

#include "cudatoolkit/gpu_facade.cuh"
#include "cudatoolkit/gpu_code_utilities.cuh"

__global__ void calcAccelerationsKernel(
		const float *masses,
		const float *positions,
		float *accelerations,
		const unsigned int numBodies,
		const float softeningFactorSquared
) {
	// the global thread ID, which is unique across all threads in the grid.
	const unsigned int globalThreadId = CudaToolkit::GpuCodeUtilities::getGlobalThreadId();
	const unsigned int xCoordinateIndexBody1 = globalThreadId * 3;
	const unsigned int yCoordinateIndexBody1 = xCoordinateIndexBody1 + 1;
	const unsigned int zCoordinateIndexBody1 = xCoordinateIndexBody1 + 2;

	float forceVector[3] = {0.0f, 0.0f, 0.0f};
	for (unsigned int i = 0; (i < numBodies) && (globalThreadId < numBodies); ++i) {
		if (globalThreadId != i) {
			const unsigned int xCoordinateIndexBody2 = i * 3;
			const unsigned int yCoordinateIndexBody2 = xCoordinateIndexBody2 + 1;
			const unsigned int zCoordinateIndexBody2 = xCoordinateIndexBody2 + 2;

			const float distanceVectorXCoordinate = positions[xCoordinateIndexBody1] - positions[xCoordinateIndexBody2];
			const float distanceVectorYCoordinate = positions[yCoordinateIndexBody1] - positions[yCoordinateIndexBody2];
			const float distanceVectorZCoordinate = positions[zCoordinateIndexBody1] - positions[zCoordinateIndexBody2];

			const float distance = sqrt(
					(distanceVectorXCoordinate * distanceVectorXCoordinate) +
					(distanceVectorYCoordinate * distanceVectorYCoordinate) +
					(distanceVectorZCoordinate * distanceVectorZCoordinate)
			) + softeningFactorSquared; // to avoid zero in the following divisions

			const float normalizedDistanceVectorXCoordinate = distanceVectorXCoordinate / distance;
			const float normalizedDistanceVectorYCoordinate = distanceVectorYCoordinate / distance;
			const float normalizedDistanceVectorZCoordinate = distanceVectorZCoordinate / distance;
			const float receivedForce = masses[i] / (distance * distance);
			// It is possible to write directly in to the accelerations array here, but the false sharing will
			// reduce in this loop significantly the performance
			forceVector[0] += (receivedForce * normalizedDistanceVectorXCoordinate);
			forceVector[1] += (receivedForce * normalizedDistanceVectorYCoordinate);
			forceVector[2] += (receivedForce * normalizedDistanceVectorZCoordinate);
		}
	}
	// false sharing is ok here
	accelerations[xCoordinateIndexBody1] =
			CudaToolkit::GpuCodeUtilities::castDoubleToFloat(GRAVITATIONAL_CONSTANT * forceVector[0]);
	accelerations[yCoordinateIndexBody1] =
			CudaToolkit::GpuCodeUtilities::castDoubleToFloat(GRAVITATIONAL_CONSTANT * forceVector[1]);
	accelerations[zCoordinateIndexBody1] =
			CudaToolkit::GpuCodeUtilities::castDoubleToFloat(GRAVITATIONAL_CONSTANT * forceVector[2]);
}

using namespace physics;

__host__ void CudaAccelerationCalculationImpl::calcAccelerations(
		const Bodies<float, float, float> &bodies,
		const size_t numBodies,
		float *accelerations,
		const float squaredSofteningFactor
) {
	const size_t floatScalarGpuBufferSize = sizeof(float) * numBodies;
	// reuse floatScalarGpuBufferSize in this calculation:
	const size_t float3dVectorGpuBufferSize = floatScalarGpuBufferSize * 3;

	CudaToolkit::GpuMemory<float> massesGpuBuffer =
			CudaToolkit::gpuFacade.allocateGpuMemory<float>(floatScalarGpuBufferSize);
	CudaToolkit::GpuMemory<float> positionsGpuBuffer =
			CudaToolkit::gpuFacade.allocateGpuMemory<float>(float3dVectorGpuBufferSize);
	CudaToolkit::GpuMemory<float> accelerationsGpuBuffer =
			CudaToolkit::gpuFacade.allocateGpuMemory<float>(float3dVectorGpuBufferSize);

	CudaToolkit::gpuFacade.copyDataFromCpuMemoryToGpuMemory<float>(bodies.masses, massesGpuBuffer,
																   floatScalarGpuBufferSize);
	CudaToolkit::gpuFacade.copyDataFromCpuMemoryToGpuMemory<float>(bodies.positions, positionsGpuBuffer,
																   float3dVectorGpuBufferSize);

	int maxSuggestedBlockSize = 0; // Suggested block size to achieve maximum occupancy.
	int unused = 0;
	hipOccupancyMaxPotentialBlockSize(
			&unused,
			&maxSuggestedBlockSize,
			calcAccelerationsKernel,
			unused,
			static_cast<int>(numBodies)
	);

	const int numThreadsPerBlock = std::min(static_cast<int>(numBodies), maxSuggestedBlockSize);
	const int numBlocks = std::ceil(static_cast<float>(numBodies) / static_cast<float>(numThreadsPerBlock));
	calcAccelerationsKernel<<<numBlocks, numThreadsPerBlock>>>(
			massesGpuBuffer,
			positionsGpuBuffer,
			accelerationsGpuBuffer,
			numBodies,
			squaredSofteningFactor
	);

	// This copy blocks until the above kernel call returns
	CudaToolkit::gpuFacade.copyDataFromGpuMemoryToCpuMemory(
			accelerationsGpuBuffer,
			accelerations,
			float3dVectorGpuBufferSize
	);
}
